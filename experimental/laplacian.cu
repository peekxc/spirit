#include "hip/hip_runtime.h"
#include <vector> 
#include <cinttypes>
 
using index_t = int64_t;
using std::vector;
using vector2D = std::vector< std::vector< index_t > >;

// Baseline from: https://stackoverflow.com/questions/44718971/calculate-binomial-coffeficient-very-reliably
// Requires O(min{k,n-k}), uses pascals triangle approach (+ degenerate cases)
__device__ inline size_t binom(size_t n, size_t k) noexcept {
  return
    (k > n) ? 0 :                  // out of range
    (k == 0 || k == n) ? 1 :       // edge
    (k == 1 || k == n-1) ? n :     // first
    (k+k < n) ?                    // recursive:
    (binom(n-1,k-1) * n)/k :       //  path to k=1   is faster
    (binom(n-1,k) * n)/(n-k);      //  path to k=n-1 is faster
}

// Table to cache low values of the binomial coefficient
// template< typename value_t = index_t >
// struct BinomialCoefficientTable {
//   size_t pre_n = 0;
//   size_t pre_k = 0; 
//   vector< vector< value_t > > BT;

//   // Safe, general binomial coefficient; uses cached table if possible 
//   value_t operator()(const index_t n, const index_t k) const {
//     if (n <= pre_n && k <= pre_k){ return BT[k][n]; } 				// runtime computed extension table
//     return static_cast< value_t >(binom(n,std::min(k,n-k)));
//   }
//   // Fast, unsafe access to a precompute table
//   constexpr auto at(index_t n, index_t k) const noexcept -> index_t {
//     return BT[k][n];
//   }

//   // Precompute a *larger* table of binomial coefficients
  void precompute(index_t n, index_t k){
    pre_n = n;
    pre_k = k;
    BT = std::vector< std::vector< value_t > >(k + 1, std::vector< value_t >(n + 1, 0));
    for (index_t i = 0; i <= n; ++i) {
      BT[0][i] = 1;
      for (index_t j = 1; j < std::min(i, k + 1); ++j){
        BT[j][i] = binom(i,j); // BT[j - 1][i - 1] + BT[j][i - 1];
      }
      if (i <= k) { BT[i][i] = 1; };
    }
  }
// }; // BinomialCoefficientTable

__device__ index_t get_max(index_t top, index_t bottom, const index_t r, const index_t m, const vector2D& BT) noexcept {
  if (!(BT[m][bottom] <= r)) { return bottom; }
  index_t size = (top - bottom);
  while (size > 0){
    index_t step = size >> 1;
    index_t mid = top - step;
    if (BT[m][mid] <= r){
    // if (!pred(mid)){
      top = mid - 1;
      size -= step + 1;
    } else {
      size = step;
    }
  }
  return top;
}

__device__ index_t get_max_vertex(const index_t r, const index_t m, const index_t n, const vector2D& BT) noexcept {
  index_t k_lb = m - 1;
  return 1 + get_max(n, k_lb, r, m, BT);
}

// Enumerates the facets on the boundary of 'simplex'
template< typename Lambda > 
__device__ void enum_boundary(const int n, const int simplex, const int dim, const vector2D& BT, Lambda&& f) {
  index_t idx_below = simplex;
  index_t idx_above = 0; 
  index_t j = n - 1;
  bool cont_enum = true; 
  for (index_t k = dim; k >= 0 && cont_enum; --k){
    j = get_max_vertex(idx_below, k + 1, j, BT) - 1; // NOTE: Danger!
    index_t c = BT[k+1][j];
    index_t face_index = idx_above - c + idx_below;
    idx_below -= c;
    idx_above += BT[k][j];
    cont_enum = f(face_index);
  }
}

__device__ void k_boundary(const index_t n, const index_t simplex, const index_t dim, const vector2D& BT, index_t* br) {
  index_t idx_below = simplex;
  index_t idx_above = 0; 
  index_t j = n - 1;
  bool cont_enum = true; 
  for (index_t k = dim; k >= 0 && cont_enum; --k){
    j = get_max_vertex(idx_below, k + 1, j, BT) - 1; // NOTE: Danger!
    index_t c = BT[k+1][j];
    index_t face_index = idx_above - c + idx_below;
    idx_below -= c;
    idx_above += BT[k][j];
    br[dim-k] = face_index;
  }
}

void compute_deg_full(const int n, const int k, const vector2D& BT, std::vector< size_t >& deg){
  for (size_t r = 0; r < BT[k][n]; ++r){
    // enum_boundary(n, r, k - 1, BT, [&](int pr){ deg[pr] += 1; return true; });
    index_t ps[3]; 
    k_boundary(n, r, k - 1, BT, ps);
    for(size_t i = 0; i < 3; ++i){
      deg[ps[i]] += 1;
    }
  }
}

// Kernel for adding the boundary/coboundary of r 
// k should == array size 
__global__ void add_boundaries(const index_t k, const index_t n, const index_t N, const index_t* sgn_pattern, const vector2D& BT, const float* x, float *y)
{
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < N){
    index_t ps[3]; 
    k_boundary(n, tid, k - 1, BT, &ps);
    int cc = 0; 
    for (int ii = 0; ii < k; ++ii){
      for (int jj = ii + 1; jj < k; ++jj){
        atomicAdd(y + ps[ii], sgn_pattern[cc] * x[ps[jj]]);
        ++cc;
      }
    }
  }  
}

void compute_deg_full(const int n, const int k, const index_t* BT, index_t* deg){
  for (size_t r = 0; r < binom(n, k); ++r){
    index_t ps[3];
    k_boundary(n, r, k - 1, BT, ps);
    for(size_t i = 0; i < 3; ++i){
      deg[ps[i]] += 1;
    }
  }
}